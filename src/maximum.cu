#include "hip/hip_runtime.h"
#include "maximum.h"

#ifdef __clang__
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_runtime_wrapper.h>
#endif

#include <math.h>
#include <stddef.h>
#include <stdio.h>
#include <time.h>

#include "util.h"

float maximum_cpu(float* v, size_t d, float* elapsed_ms) {
    float max = log(0.f); // negative infinity
    size_t len = 1 << d;  // Calcul de la taille à partir de la dimension de l'hypercube

    clock_t start = clock();
    for (size_t idx = 0; idx < len; idx++)
        max = MAX(max, v[idx]);
    clock_t end = clock();

    *elapsed_ms = 1000.f * (float)(end - start) / CLOCKS_PER_SEC;

    return max;
}

// Inline function that swap two pointers poiting to buffers
__device__ __host__ inline void swap(void** lhs, void** rhs) {
    void* tmp = *lhs;
    *lhs = *rhs;
    *rhs = tmp;
}

__global__ void maximum_dim_seq_kernel(float* read, float* write, int i) {
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;

    size_t neightbour_mask = 1 << i;
    write[idx] = MAX(read[idx], read[idx ^ neightbour_mask]);
}

// Calcul atomiquement (un seul appel de kernel) le maximum d'un tableau en le
// préchargeant en mémoire partagée
__global__ void maximum_atomic(float* read, float* write, int d) {
    __shared__ float A[MAX_THREADS_PER_BLOCK];
    __shared__ float B[MAX_THREADS_PER_BLOCK];

    unsigned int threadId = threadIdx.x;

    // Chargement en mémoire partagée
    A[threadId] = read[threadId];
    __syncthreads();

    // Calcul du maximum
    float* R = A;
    float* W = B;

    size_t neightbour_mask = 1;
    for (int i = 0; i < d; i++) {
        // Calcul
        W[threadId] = MAX(R[threadId ^ neightbour_mask], R[threadId]);
        neightbour_mask <<= 1;
        // Echange des buffers
        swap((void**)&R, (void**)&W);
        // Syncronisation
        __syncthreads();
    }

    // Ecriture en mémoire centrale
    write[threadId] = R[threadId];
}

float maximum_dim_seq(float* h_v, size_t d, float* elapsed_ms) {
    // Gestion des erreurs
    hipError_t err = hipSuccess;

    // Mesure temporelle
    hipEvent_t start = 0, end = 0;

    // Mémoire GPU
    float *d_A, *d_B;
    size_t len = 1 << d;
    size_t bytes = sizeof(float) * len;

    // Création des événements
    err = hipEventCreate(&start);
    EXPECT(err, "Could not create start event");

    err = hipEventCreate(&end);
    EXPECT(err, "Could not create start end");

    // Allocation memoire GPU
    err = hipMalloc((void**)&d_A, bytes);
    EXPECT(err, "Could not allocate d_A on device memory");

    err = hipMalloc((void**)&d_B, bytes);
    EXPECT(err, "Could not allocate d_B on device memory");

    // Copie des données CPU vers GPU
    err = hipMemcpy(d_A, h_v, bytes, hipMemcpyHostToDevice);
    EXPECT(err, "Could not copy h_A to d_A");

    // Execution du kernel
    size_t threadsPerBlock = MIN(MAX_THREADS_PER_BLOCK, len);
    size_t blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
    dim3 dimBlocks(blocksPerGrid, 1, 1);
    dim3 dimThreads(threadsPerBlock, 1, 1);

    // Exécution du kernel
    err = hipEventRecord(start);
    EXPECT(err, "Couldn't not record start event");

    float* d_read = d_A;
    float* d_write = d_B;
    bool swapped = false;
    if (dimBlocks.x == 1) {
        maximum_atomic<<<dimBlocks, dimThreads>>>(d_read, d_write, d);
    } else {
        for (int i = 0; i < d; i++) {
            // Calcul du noyau pour la dimension i
            maximum_dim_seq_kernel<<<dimBlocks, dimThreads>>>(d_read, d_write, i);
            // Echange des buffers
            swap((void**)&d_read, (void**)&d_write);
            swapped = !swapped;
            // Syncronisation
            err = hipDeviceSynchronize();
            EXPECT(err, "Error while synchronizing device");
        }
    }

    err = hipGetLastError();
    EXPECT(err, "Error while executing kernel on GPU");

    err = hipEventRecord(end);
    EXPECT(err, "Couln't not record end event");

    // Récupération tps d'exécution
    err = hipEventSynchronize(end);
    EXPECT(err, "Error while syncronizing end event");

    err = hipEventElapsedTime(elapsed_ms, start, end);
    EXPECT(err, "Error while measuring elapsed time");

    // Copie de vecteur d_A/B[0] vers h_max
    float h_max = 0;
    if (dimBlocks.x == 1) {
        err = hipMemcpy(&h_max, d_write, sizeof(float), hipMemcpyDeviceToHost);
    } else if (swapped) {
        err = hipMemcpy(&h_max, d_B, sizeof(float), hipMemcpyDeviceToHost);
    } else {
        err = hipMemcpy(&h_max, d_A, sizeof(float), hipMemcpyDeviceToHost);
    }
    EXPECT(err, "Could not copy d_A/d_B[0] to h_max [swapped = %d]", swapped);

    return h_max;
}
